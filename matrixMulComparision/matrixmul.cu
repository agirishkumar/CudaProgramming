#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

// CUDA Kernel for Matrix Multiplication
__global__ void matrixMul(const int *A, const int *B, int *C, int m, int n, int p) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < m && col < p) {
        int sum = 0;
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B[k * p + col];
        }
        C[row * p + col] = sum;
    }
}

void initializeMatrix(int *mat, int rows, int cols) {
    int value = 0;
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            mat[i * cols + j] = ++value; // Sequential integers
        }
    }
}

int main() {
    int m = 512, n = 512, p = 512;
    size_t sizeA = m * n * sizeof(int);
    size_t sizeB = n * p * sizeof(int);
    size_t sizeC = m * p * sizeof(int);

    int *h_A = new int[m * n];
    int *h_B = new int[n * p];
    int *h_C = new int[m * p];
    
    // Initialize matrices
    initializeMatrix(h_A, m, n);
    initializeMatrix(h_B, n, p);

    int *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((p + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (m + threadsPerBlock.y - 1) / threadsPerBlock.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    matrixMul<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, m, n, p);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "CUDA execution time: " << milliseconds << " ms\n";

    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);

    // Display the first element for verification
    std::cout << "First element of result (CUDA): " << h_C[0] << std::endl;

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
