
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

// CUDA Kernel function to multiply the elements of two arrays
__global__ void multiply(int *a, int *b, int *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        c[index] = a[index] * b[index];
    }
}

void random_ints(int* x, int size) {
    for (int i = 0; i < size; i++) {
        x[i] = rand() % 100;
    }
}

int main() {
    int n = 1024; // number of elements in each array
    int *a, *b, *c;           // host copies of a, b, c
    int *d_a, *d_b, *d_c;     // device copies of a, b, c
    int size = n * sizeof(int);

    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Setup input values
    a = (int *)malloc(size); random_ints(a, n);
    b = (int *)malloc(size); random_ints(b, n);
    c = (int *)malloc(size);

    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch multiply() kernel on GPU with enough blocks
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    multiply<<<numBlocks, blockSize>>>(d_a, d_b, d_c, n);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Display the results
    for(int i = 0; i < 10; i++) {
        std::cout << a[i] << "*" << b[i] << "=" << c[i] << std::endl;
    }

    // Cleanup
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}
